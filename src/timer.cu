#include "timer.cuh"

__host__ hipEvent_t get_time()
{
    hipEvent_t time;
    hipEventCreate(&time);
    hipEventRecord(time);
    return time;
}

__host__ float get_delta(hipEvent_t start, hipEvent_t stop)
{
    hipEventSynchronize(stop);

    float delta = 0;
    hipEventElapsedTime(&delta, start, stop);
    return delta;
}
