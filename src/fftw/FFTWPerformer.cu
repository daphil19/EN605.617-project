#include "hip/hip_runtime.h"
#include "FFTWPerformer.cuh"

#include <cmath>
#include <memory>
#include <string>
#include <algorithm>
#include <complex>

#include <fftw3.h>
#include <AudioFile.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>

#include "../io/SampleSource.h"
#include "../hann.cuh"
#include "CPUSamples.cuh"

// TODO maybe the interface is a spectrogramPerformer that has a function that returns a resultant 2-d double array

// this is value is smaller than the log10 of the smallest positive double value
#define MIN_REPLACEMENT -350.0


FFTWPerformer::FFTWPerformer(int fft_size, const std::string file)
{
    this->fft_size = fft_size;
    window = hann<thrust::host_vector<double> >(fft_size);

    // So, this might not work because of the fact that all of the data gets loaded into memory...
    // but the fact that it is a vector means we might be able to get away with a bunch of data? idk...
    source = AudioFile<double>(file);

    // TODO we might not need complex as a member if we can get away with a void pointer that returns the samples naiively
    complex = source.getNumChannels() == 2;

    // this internal output buffer is used so that we can have a single plan, but also emit defensive copies of windows when the code gets executed
    out_buffer = fftw_alloc_complex(fft_size);


    in_buffer = std::unique_ptr<CPUSamples>(new CPUSamples(complex, fft_size));

    window_step_size = fft_size / 2;

    // TODO we might be able to simplify this by providing a void pointer?
    plan = complex ? fftw_plan_dft_1d(fft_size, in_buffer->getComplex(), out_buffer, FFTW_FORWARD, FFTW_ESTIMATE): fftw_plan_dft_r2c_1d(fft_size, in_buffer->getReal(), out_buffer, FFTW_ESTIMATE);
    
}

FFTWPerformer::~FFTWPerformer()
{
    fftw_destroy_plan(plan);
    fftw_free(out_buffer);
}

void FFTWPerformer::performFFT() {

    // TODO is it a window and then an fft, or an fft and then a window?


    // for every fftsize, we will step fftsize / 2
    // we repeat this until we have an fftsize that contains the end of the file

    auto num_samples = source.getNumSamplesPerChannel();


    // the number of colums in the output
    int num_cols = (num_samples / (fft_size / 2)) ;


    // TODO better typing

    // in this structure, the outer index is the column of the image, and the inner index is the row
    // this allows for fft outputs to be bulk copied via memcpy as opposed to iterated over
    // hopefully, we don't need to transpose to get things to work later on

    // TODO WE SHOULD IMPROVE WHAT THE OUTPUT STRUCTURE IS!
    // TODO do we want to convert this to an array? or maybe have it be a thrust vector??
    // std::unique_ptr<std::unique_ptr<double[]>[]> output(new std::unique_ptr<double[]>[num_cols]);

    // TODO will making these references make things faster?
    thrust::host_vector<thrust::host_vector<double> >output(num_cols);

    for (int i = 0; i < num_cols; i++) {
        // first, allocate the results we will be using
        auto cur_col = thrust::host_vector<double>(output_fft_size);
        

        // clear the input buffer in the event we don't have enough data to fill the buffer
        in_buffer->clear();

        // TODO next, we need to load data into the buffer

        // TODO is there a better way for this to be done?
        auto start = fft_size / 2 * i;
        auto end = std::min(start + fft_size, source.getNumSamplesPerChannel());

        // now we _actually_ load the samples
        // source.samples
        in_buffer->load(source.samples, start, end);

        // TODO WE NEED TO DO THINKS LIKE NORMALIZE!

        // window
        in_buffer->applyWindow(window);

        // execute
        fftw_execute(plan);

        // next, we take the magnitude
        // can we get away with shifting to make this faster?

        // write data back to output 
        // TODO how are we supposed to get back to a single data type? is that by taking the magnitude?


        // casting to complex array helps with normalization
        auto out_buf_cast = reinterpret_cast<std::complex<double> *>(out_buffer);
        
        // copy contents into the output, getting the magnitude along the way
        // TODO, also, don't forget about zero samples! this will result in a NaN
        // do we want to just make those the smallest positive double value?
        for (int j = 0; j < cur_col.size(); j++) {
            cur_col[j] = std::abs(out_buf_cast[j]);
            // TODO do we want to handle the zero case here?
        }

        // log scale... 10 log 10
        // thrust::transform(cur_col.begin(), cur_col.end(), thrust::make_constant_iterator(10.0), cur_col.begin(), thrust::multiplies<double>());

        // TODO log
        // TODO do i need host here?
        // log scale... 10 log 10
        thrust::transform(cur_col.begin(), cur_col.end(), cur_col.begin(), [=] (double x) {
            double logscale = 10.0 * log10(x);
            if (isfinite(logscale)) {
                logscale = MIN_REPLACEMENT;
            }
            return logscale;
        });


        // put the results in the output
        output[i] = cur_col;

    }

    /*
        NOTE I think this is the intended order:
        get samples
        normalize
        window
        fft
        results go into pixel
        slide by ...?
     */


}
