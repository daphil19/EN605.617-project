#include "hip/hip_runtime.h"
#include "FFTWPerformer.cuh"

#include <cmath>
#include <memory>
#include <string>
#include <algorithm>
#include <complex>

#include <iostream>

#include <fftw3.h>
#include <AudioFile.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/execution_policy.h>

#include "../io/SampleSource.h"
#include "../hann.cuh"
#include "CPUSamples.cuh"

// TODO maybe the interface is a spectrogramPerformer that has a function that returns a resultant 2-d double array

// this is value is smaller than the log10 of the smallest positive double value
#define MIN_REPLACEMENT -350.0


FFTWPerformer::FFTWPerformer(int fft_size, const std::string file)
{
    this->fft_size = fft_size;
    window = hann<thrust::host_vector<double> >(fft_size);

    // So, this might not work because of the fact that all of the data gets loaded into memory...
    // but the fact that it is a vector means we might be able to get away with a bunch of data? idk...
    source = AudioFile<double>(file);

    // TODO we might not need complex as a member if we can get away with a void pointer that returns the samples naiively
    complex = source.getNumChannels() == 2;

    // this internal output buffer is used so that we can have a single plan, but also emit defensive copies of windows when the code gets executed
    out_buffer = fftw_alloc_complex(fft_size);


    in_buffer = std::unique_ptr<CPUSamples>(new CPUSamples(complex, fft_size));

    window_step_size = fft_size / 2;

    // TODO we might be able to simplify this by providing a void pointer?
    plan = complex ? fftw_plan_dft_1d(fft_size, in_buffer->getComplex(), out_buffer, FFTW_FORWARD, FFTW_ESTIMATE): fftw_plan_dft_r2c_1d(fft_size, in_buffer->getReal(), out_buffer, FFTW_ESTIMATE);
    
    output_fft_size = complex ? fft_size : (fft_size / 2) + 1;


}

FFTWPerformer::~FFTWPerformer()
{
    fftw_destroy_plan(plan);
    fftw_free(out_buffer);
}

void FFTWPerformer::normalize() {
    double step = 1.0 / source.getBitDepth();
    int offset = pow(2, source.getBitDepth() - 1) + 1; // intervals are usually [-2^(n - 1) - 1, 2^(n - 1)] for data types

    
}

void FFTWPerformer::performFFT() {


    std::cout << "fftw performer will use up to " << fftw_planner_nthreads() << " threads" << std::endl;

    // TODO is it a window and then an fft, or an fft and then a window?


    // for every fftsize, we will step fftsize / 2
    // we repeat this until we have an fftsize that contains the end of the file

    auto num_samples = source.getNumSamplesPerChannel();


    // the number of colums in the output
    // we are aiming for a 50% overlap
    int num_cols = num_samples / (fft_size / 2);


    // TODO better typing

    // in this structure, the outer index is the column of the image, and the inner index is the row
    // this allows for fft outputs to be bulk copied via memcpy as opposed to iterated over
    // hopefully, we don't need to transpose to get things to work later on

    // TODO WE SHOULD IMPROVE WHAT THE OUTPUT STRUCTURE IS!
    // TODO do we want to convert this to an array? or maybe have it be a thrust vector??
    // std::unique_ptr<std::unique_ptr<double[]>[]> output(new std::unique_ptr<double[]>[num_cols]);

    // TODO will making these references make things faster?
    thrust::host_vector<thrust::host_vector<double> >output(num_cols);

    for (int i = 0; i < num_cols; i++) {
        // std::cout << i << std::endl;


        // first, allocate the results we will be using
        auto cur_col = thrust::host_vector<double>(output_fft_size);
        

        // clear the input buffer in the event we don't have enough data to fill the buffer
        in_buffer->clear();

        // TODO next, we need to load data into the buffer

        // TODO is there a better way for this to be done?
        // these numbers are in samples!
        auto start = fft_size / 2 * i;
        auto end = std::min(start + fft_size, source.getNumSamplesPerChannel());

        // now we _actually_ load the samples
        // source.samples
        in_buffer->load(source.samples, start, end);

        in_buffer->normalize(source.getBitDepth());

        // TODO WE NEED TO DO THINKS LIKE NORMALIZE!

        // window
        in_buffer->applyWindow(window);

        // execute
        fftw_execute(plan);

        // next, we take the magnitude
        // can we get away with shifting to make this faster?

        // write data back to output 
        // TODO how are we supposed to get back to a single data type? is that by taking the magnitude?


        // casting to complex array helps with normalization
        auto out_buf_cast = reinterpret_cast<std::complex<double> *>(out_buffer);
        
        // std::cout << cur_col.size() << std::endl;
        // copy contents into the output, getting the magnitude along the way
        // TODO, also, don't forget about zero samples! this will result in a NaN
        // do we want to just make those the smallest positive double value?

        // TODO would it be possible for this to be something we do in the transform below as well?
        // for (int j = 0; j < cur_col.size(); j++) {
        //     double magSquared = pow(std::abs(out_buf_cast[j]), 2);
        //     double logScale = 10.0 * log10(magSquared); 
        //     cur_col[j] = isfinite(logScale) ? logScale : MIN_REPLACEMENT;
        // }
        

        // while this would leverage a thrust transform, the parallel benefit from thrust doesn't outweigh the double-iteration cost
        // the for loop above that iteratively transforms the data and stores it in the results array is faster
        // also, trying to wrap the pointer in a vector and doing the transform all together is even slower than this!
        
        // this one seems to be the fastest, as we don't have an extra copy and still benefit from some of the transform being parallel
        for (int j = 0; j < cur_col.size(); j++) {
            cur_col[j] = std::abs(out_buf_cast[j]);
        }
        thrust::transform(cur_col.begin(), cur_col.end(), cur_col.begin(), [=] (double x) {
            double logscale = 10.0 * log10(pow(x, 2));
            if (!isfinite(logscale)) {
                logscale = MIN_REPLACEMENT;
            }
            return logscale;
        });


        // this approach may be the slowest of the three, all things considered
        // BUT, with the optimizations turned on it actually ends up being pretty fast...
        // auto out_buf_vec = thrust::host_vector<std::complex<double>>(out_buf_cast, out_buf_cast + output_fft_size);
        // thrust::transform(out_buf_vec.begin(), out_buf_vec.end(), cur_col.begin(), [=] (std::complex<double> x) {
        //     double magSquared = pow(std::abs(x), 2);
        //     double logScale = 10.0 * log10(magSquared);
        //     return isfinite(logScale) ? logScale : MIN_REPLACEMENT;
        // });
        


        // put the results in the output
        output[i] = cur_col;
        // for (int j = 0; j < output_fft_size; j++) {
        //     std::cout << cur_col[j] << " ";
        // }
        // std::cout << "\\" << std::endl;

    }

    /*
        NOTE I think this is the intended order:
        get samples
        normalize
        window
        fft
        results go into pixel
        slide by ...?
     */


}
