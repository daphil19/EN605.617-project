#include "hip/hip_runtime.h"
#include <iostream>

#include <string>
#include <vector>
#include <complex>

#include <AudioFile.h>
#include <thrust/host_vector.h>
#include <thrust/extrema.h>
#include <thrust/sequence.h>

#include "cuda_utilities.cuh"
#include "hann.cuh"
#include "fftw/FFTWPerformer.cuh"
#include "cuda/CUFFTPerformer.cuh"
#include <chrono>
#include <thread>

// adapted from https://stackoverflow.com/a/47785639
const int BYTES_PER_PIXEL = 3; /// red, green, & blue
const int FILE_HEADER_SIZE = 14;
const int INFO_HEADER_SIZE = 40;

// i could (maybe should?) refactor all of the unsigned char* refs to std::byte, but not right now
unsigned char *createBitmapFileHeader(int height, int stride)
{
    int fileSize = FILE_HEADER_SIZE + INFO_HEADER_SIZE + (stride * height);

    static unsigned char fileHeader[] = {
        0, 0,       /// signature
        0, 0, 0, 0, /// image file size in bytes
        0, 0, 0, 0, /// reserved
        0, 0, 0, 0, /// start of pixel array
    };

    fileHeader[0] = (unsigned char)('B');
    fileHeader[1] = (unsigned char)('M');
    fileHeader[2] = (unsigned char)(fileSize);
    fileHeader[3] = (unsigned char)(fileSize >> 8);
    fileHeader[4] = (unsigned char)(fileSize >> 16);
    fileHeader[5] = (unsigned char)(fileSize >> 24);
    fileHeader[10] = (unsigned char)(FILE_HEADER_SIZE + INFO_HEADER_SIZE);

    return fileHeader;
}

unsigned char *createBitmapInfoHeader(int height, int width)
{
    static unsigned char infoHeader[] = {
        0, 0, 0, 0, /// header size
        0, 0, 0, 0, /// image width
        0, 0, 0, 0, /// image height
        0, 0,       /// number of color planes
        0, 0,       /// bits per pixel
        0, 0, 0, 0, /// compression
        0, 0, 0, 0, /// image size
        0, 0, 0, 0, /// horizontal resolution
        0, 0, 0, 0, /// vertical resolution
        0, 0, 0, 0, /// colors in color table
        0, 0, 0, 0, /// important color count
    };

    infoHeader[0] = (unsigned char)(INFO_HEADER_SIZE);
    infoHeader[4] = (unsigned char)(width);
    infoHeader[5] = (unsigned char)(width >> 8);
    infoHeader[6] = (unsigned char)(width >> 16);
    infoHeader[7] = (unsigned char)(width >> 24);
    infoHeader[8] = (unsigned char)(height);
    infoHeader[9] = (unsigned char)(height >> 8);
    infoHeader[10] = (unsigned char)(height >> 16);
    infoHeader[11] = (unsigned char)(height >> 24);
    infoHeader[12] = (unsigned char)(1);
    infoHeader[14] = (unsigned char)(BYTES_PER_PIXEL * 8);

    return infoHeader;
}

void generateBitmapImage(unsigned char *image, int height, int width, char *imageFileName)
{
    int widthInBytes = width * BYTES_PER_PIXEL;

    unsigned char padding[3] = {0, 0, 0};
    int paddingSize = (4 - (widthInBytes) % 4) % 4;

    int stride = (widthInBytes) + paddingSize;

    FILE *imageFile = fopen(imageFileName, "wb");

    unsigned char *fileHeader = createBitmapFileHeader(height, stride);
    fwrite(fileHeader, 1, FILE_HEADER_SIZE, imageFile);

    unsigned char *infoHeader = createBitmapInfoHeader(height, width);
    fwrite(infoHeader, 1, INFO_HEADER_SIZE, imageFile);

    int i;
    for (i = 0; i < height; i++)
    {
        fwrite(image + (i * widthInBytes), BYTES_PER_PIXEL, width, imageFile);
        fwrite(padding, 1, paddingSize, imageFile);
    }

    fclose(imageFile);
}

void outputResultsToFile(thrust::host_vector<thrust::host_vector<double>> const &results, char *outfile)
{
    thrust::host_vector<double> maxs(results.size());
    thrust::transform(results.begin(), results.end(), maxs.begin(), [=](thrust::host_vector<double> column)
                      { return *thrust::max_element(column.begin(), column.end()); });
    thrust::host_vector<double> mins(results.size());
    thrust::transform(results.begin(), results.end(), mins.begin(), [=](thrust::host_vector<double> column)
                      { return *thrust::min_element(column.begin(), column.end()); });

    double maxOfMaxs = *thrust::max_element(maxs.begin(), maxs.end());
    double maxOfMins = *thrust::max_element(mins.begin(), mins.end());

    size_t height = results[0].size();
    size_t width = results.size();

    thrust::host_vector<thrust::host_vector<unsigned char>> bytes(results.size(), thrust::host_vector<unsigned char>(results[0].size()));

    for (int i = 0; i < results.size(); i++)
    {
        thrust::transform(results[i].begin(), results[i].end(), bytes[i].begin(), [=](double res)
                          {
            if (res <= maxOfMins) {
                return (unsigned char)255;
            } else if (res >= maxOfMaxs) {
                return (unsigned char)0;
            } else {
                return (unsigned char)round( 255 * (res - maxOfMins) / (maxOfMaxs - maxOfMins));
            } });
    }

    unsigned char imageBytes[height][width][BYTES_PER_PIXEL];
    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            for (int k = 0; k < BYTES_PER_PIXEL; k++)
            {
                imageBytes[i][j][k] = bytes[j][i];
            }
        }
    }

    generateBitmapImage((unsigned char *)imageBytes, height, width, outfile);
}

void verifySpectrogramOutputs()
{
    int fft_size = 256;

    // auto filename = "../../testing123-mono.wav";
    AudioFile<double> source("../../testing123-mono.wav");
    // TODO consider just passing in a reference to the source?
    FFTWPerformer fftw(fft_size, source);
    auto fftwResults = fftw.performFFT();
    outputResultsToFile(fftwResults, (char *)"../../fftw-results.bmp");

    CUFFTPerformer cufft(fft_size, source);
    auto cufftResults = cufft.performFFT();
    outputResultsToFile(fftwResults, (char *)"../../cufft-results.bmp");
}

void performBenchmark()
{
    AudioFile<double> source("../../sermon.wav");
    for (int i = 8; i <= 20; i++) {
        int fft_size = pow(2, i);
        std::cout << "using fft size " << fft_size << std::endl;
        FFTWPerformer p(fft_size, source);
        std::cout << "Successfuly loaded!" << std::endl;
        std::chrono::steady_clock::time_point fftwBegin = std::chrono::steady_clock::now();
        auto results = p.performFFT();
        std::chrono::steady_clock::time_point fftwEnd = std::chrono::steady_clock::now();
        std::cout << "done cpu in: " << std::chrono::duration_cast<std::chrono::milliseconds>(fftwEnd - fftwBegin).count() << std::endl;

        CUFFTPerformer p2(fft_size, source);
        std::cout << "Beginning the gpu one..." << std::endl;
        hipEvent_t cufftStart = get_time();
        auto results2 = p2.performFFT();
        hipEvent_t cufftEnd = get_time();
        std::cout << "done gpu in: " << get_delta(cufftStart, cufftEnd) << std::endl;
    }
    // fft sizes should range from 256 to... something? maybe 2^20?
    // int fft_size = 8192;

}

int main(int argc, char const *argv[])
{
    bool verify = argc == 2 && std::string("--verify").compare(argv[1]) == 0;

    fftw_init_threads();
    fftw_plan_with_nthreads(std::thread::hardware_concurrency());

    if (verify)
    {
        std::cout << "Running spectrogram verification code paths." << std::endl;
        verifySpectrogramOutputs();
    }
    else
    {
        std::cout << "Performing benchmark" << std::endl;
        performBenchmark();
    }

    fftw_cleanup();
    fftw_cleanup_threads();

    std::cout << "Done!" << std::endl;
    return EXIT_SUCCESS;
}
