#include "hip/hip_runtime.h"
#include <iostream>

#include <string>
#include <vector>
#include <complex>

#include <AudioFile.h>
#include <thrust/host_vector.h>
#include <thrust/extrema.h>
#include <thrust/sequence.h>

#include "cuda_utilities.cuh"
#include "hann.cuh"
#include "fftw/FFTWPerformer.cuh"
#include "cuda/CUFFTPerformer.cuh"
#include <chrono>
#include <thread>


// adapted from https://stackoverflow.com/a/47785639
const int BYTES_PER_PIXEL = 3; /// red, green, & blue
const int FILE_HEADER_SIZE = 14;
const int INFO_HEADER_SIZE = 40;

// void generateBitmapImage(std::byte* image, int height, int width, char* imageFileName);
// unsigned char* createBitmapFileHeader(int height, int stride);
// unsigned char* createBitmapInfoHeader(int height, int width);

// i could (maybe should?) refactor all of the unsigned char* refs to std::byte, but not right now
unsigned char* createBitmapFileHeader (int height, int stride)
{
    int fileSize = FILE_HEADER_SIZE + INFO_HEADER_SIZE + (stride * height);

    static unsigned char fileHeader[] = {
        0,0,     /// signature
        0,0,0,0, /// image file size in bytes
        0,0,0,0, /// reserved
        0,0,0,0, /// start of pixel array
    };

    fileHeader[ 0] = (unsigned char)('B');
    fileHeader[ 1] = (unsigned char)('M');
    fileHeader[ 2] = (unsigned char)(fileSize      );
    fileHeader[ 3] = (unsigned char)(fileSize >>  8);
    fileHeader[ 4] = (unsigned char)(fileSize >> 16);
    fileHeader[ 5] = (unsigned char)(fileSize >> 24);
    fileHeader[10] = (unsigned char)(FILE_HEADER_SIZE + INFO_HEADER_SIZE);

    return fileHeader;
}

unsigned char* createBitmapInfoHeader (int height, int width)
{
    static unsigned char infoHeader[] = {
        0,0,0,0, /// header size
        0,0,0,0, /// image width
        0,0,0,0, /// image height
        0,0,     /// number of color planes
        0,0,     /// bits per pixel
        0,0,0,0, /// compression
        0,0,0,0, /// image size
        0,0,0,0, /// horizontal resolution
        0,0,0,0, /// vertical resolution
        0,0,0,0, /// colors in color table
        0,0,0,0, /// important color count
    };

    infoHeader[ 0] = (unsigned char)(INFO_HEADER_SIZE);
    infoHeader[ 4] = (unsigned char)(width      );
    infoHeader[ 5] = (unsigned char)(width >>  8);
    infoHeader[ 6] = (unsigned char)(width >> 16);
    infoHeader[ 7] = (unsigned char)(width >> 24);
    infoHeader[ 8] = (unsigned char)(height      );
    infoHeader[ 9] = (unsigned char)(height >>  8);
    infoHeader[10] = (unsigned char)(height >> 16);
    infoHeader[11] = (unsigned char)(height >> 24);
    infoHeader[12] = (unsigned char)(1);
    infoHeader[14] = (unsigned char)(BYTES_PER_PIXEL*8);

    return infoHeader;
}

void generateBitmapImage (unsigned char* image, int height, int width, char* imageFileName)
{
    int widthInBytes = width * BYTES_PER_PIXEL;

    unsigned char padding[3] = {0, 0, 0};
    int paddingSize = (4 - (widthInBytes) % 4) % 4;

    int stride = (widthInBytes) + paddingSize;

    FILE* imageFile = fopen(imageFileName, "wb");

    unsigned char* fileHeader = createBitmapFileHeader(height, stride);
    fwrite(fileHeader, 1, FILE_HEADER_SIZE, imageFile);

    unsigned char* infoHeader = createBitmapInfoHeader(height, width);
    fwrite(infoHeader, 1, INFO_HEADER_SIZE, imageFile);

    int i;
    for (i = 0; i < height; i++) {
        fwrite(image + (i*widthInBytes), BYTES_PER_PIXEL, width, imageFile);
        fwrite(padding, 1, paddingSize, imageFile);
    }

    fclose(imageFile);
}

void outputResultsToFile(thrust::host_vector<thrust::host_vector<double> > const& results, char* outfile) {
    thrust::host_vector<double> maxs(results.size());
    thrust::transform(results.begin(), results.end(), maxs.begin(), [=] (thrust::host_vector<double> column) {
        return *thrust::max_element(column.begin(), column.end());
    });
    thrust::host_vector<double> mins(results.size());
    thrust::transform(results.begin(), results.end(), mins.begin(), [=] (thrust::host_vector<double> column) {
        return *thrust::min_element(column.begin(), column.end());
    });

    double maxOfMaxs = *thrust::max_element(maxs.begin(), maxs.end());
    double maxOfMins = *thrust::max_element(mins.begin(), mins.end());

    size_t height = results[0].size();
    size_t width = results.size();

    thrust::host_vector<thrust::host_vector<unsigned char> >bytes(results.size(), thrust::host_vector<unsigned char>(results[0].size()));

    for (int i = 0; i < results.size(); i++) {
        thrust::transform(results[i].begin(), results[i].end(), bytes[i].begin(), [=] (double res) {
            if (res <= maxOfMins) {
                return (unsigned char)255;
            } else if (res >= maxOfMaxs) {
                return (unsigned char)0;
            } else {
                return (unsigned char)round( 255 * (res - maxOfMins) / (maxOfMaxs - maxOfMins));
            }
        });        
    }

    unsigned char imageBytes[height][width][BYTES_PER_PIXEL];
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            for (int k = 0; k < BYTES_PER_PIXEL; k++) {
                imageBytes[i][j][k] = bytes[j][i];
            }
        }
    }

    generateBitmapImage((unsigned char*) imageBytes, height, width, outfile);

}

void verifySpectrogramOutputs() {
    int fft_size = 256;
    
    auto filename = "../../testing123-mono.wav";
    // TODO consider just passing in a reference to the source?
    FFTWPerformer fftw(fft_size, filename);
    auto fftwResults = fftw.performFFT();
    outputResultsToFile(fftwResults, (char*)"../../fftw-results.bmp");

    CUFFTPerformer cufft(fft_size, filename);
    auto cufftResults = cufft.performFFT();
    outputResultsToFile(fftwResults, (char*)"../../cufft-results.bmp");
}

int main(int argc, char const *argv[])
{
    int fft_size = 256;

    // the max size we can have for a file (on my 3080) is: 384307168202282325 samples (real)

    // TODO we will need to add more arguments (fftsize, etc.)
    // std::string filename = argc == 2 ? argv[2] : "../../testing123-mono.wav"; // TODO args!

    std::string filename = "../../testing123-mono.wav";

    bool verify = argc == 2 && argv[1] == "--verify";

    // TODO would it make more sense for me to pass around an instance of the source instead of just the filename

    // give fftw the best shot possible
    fftw_init_threads();
    fftw_plan_with_nthreads(std::thread::hardware_concurrency());

    if (verify) {
        std::cout << "Running spectrogram verification code paths." << std::endl;
        verifySpectrogramOutputs();
    } else {
        std::cout << "Performing benchmark" << std::endl;
    }

    // std::cout << "Attempting to load WAV file " << filename << "... this may take a while" << std::endl;
    // FFTWPerformer p(fft_size, filename);
    // std::cout << "Successfuly loaded!" << std::endl;
    // // hipEvent_t fftwStart = get_time();
    // std::chrono::steady_clock::time_point fftwBegin = std::chrono::steady_clock::now();
    // auto results = p.performFFT();
    // // hipEvent_t fftwEnd = get_time();
    // std::chrono::steady_clock::time_point fftwEnd = std::chrono::steady_clock::now();
    // std::cout << "done cpu in: " << std::chrono::duration_cast<std::chrono::milliseconds>(fftwEnd - fftwBegin).count() << std::endl;

    // CUFFTPerformer p2(fft_size, filename);
    // std::cout << "Beginning the gpu one..." << std::endl;
    // hipEvent_t cufftStart = get_time();
    // auto results2 = p2.performFFT();
    // hipEvent_t cufftEnd = get_time();
    // std::cout << "done gpu in: " << get_delta(cufftStart, cufftEnd) << std::endl;

    // std::cout << "Done!" << std::endl;

    fftw_cleanup();
    fftw_cleanup_threads();


    // outputResultsToFile(results2);
    return EXIT_SUCCESS;
}

/*
0-256
128-384
256-512
384-640
512-768
640-896
768-eof
*/