#include "cuda_utilities.cuh"

__host__ hipEvent_t get_time()
{
    hipEvent_t time;
    hipEventCreate(&time);
    hipEventRecord(time);
    return time;
}

__host__ float get_delta(hipEvent_t start, hipEvent_t stop)
{
    hipEventSynchronize(stop);

    float delta = 0;
    hipEventElapsedTime(&delta, start, stop);
    return delta;
}

__device__ unsigned int get_thread_index()
{
    return (blockIdx.x * blockDim.x) + threadIdx.x;
}
