#include "hip/hip_runtime.h"
#include "GPUSamples.cuh"

#include <thrust/host_vector.h>

#include "kernel_utils.cuh"

// TODO: maybe make these complex kernels a 2-d thing?

__global__ void clear_real(hipfftDoubleReal* data) {
    auto idx = get_thread_index();
    data[idx] = 0;
}

__global__ void clear_complex(hipfftDoubleComplex* data) {
    auto idx = get_thread_index();
    data[idx].x = 0;
    data[idx].y = 0;
}

__global__ void apply_window_real(hipfftDoubleReal* data, double* window) {
    auto idx = get_thread_index();
    data[idx] *= window[idx];
}

__global__ void apply_window_complex(hipfftDoubleComplex* data, double* window) {
    auto idx = get_thread_index();
    // auto sample = data[idx];
    data[idx].x *= window[idx];
    data[idx].y *= window[idx];
}

__global__ void normalize_real(hipfftDoubleReal* data, double step, int offset) {
    auto idx = get_thread_index();
    data[idx] = (data[idx] + offset) * step;
}

__global__ void normalize_complex(hipfftDoubleComplex* data, double step, int offset) {
    auto idx = get_thread_index();
    auto sample = data[idx];
    data[idx].y = (sample.y + offset) * step;
    data[idx].x = (sample.x + offset) * step;
}

GPUSamples::GPUSamples(bool complex, int fft_size)
{
    this->complex = complex;
    size = fft_size;

    // TODO consider malloc host
    // TODO we probably need to fix fft_size
    if (complex) {
        hipHostMalloc((void**) &samples.complex, fft_size * sizeof(hipfftDoubleComplex));
    } else {
        hipHostMalloc((void**) &samples.real, fft_size * sizeof(hipfftDoubleReal));
    }
}

GPUSamples::~GPUSamples()
{
    if (complex) {
        hipHostFree(samples.complex);
    } else {
        hipHostFree(samples.real);
    }
}

bool GPUSamples::isComplex() {
    return complex;
}

GPUSamples::Samples GPUSamples::getSamples() {
    return samples;
}

hipfftDoubleReal* GPUSamples::getReal() {
    return samples.real;
}

hipfftDoubleComplex* GPUSamples::getComplex() {
    return samples.complex;
}

void GPUSamples::clear() {
    // TODO WE NEED TO TUNE THE BLOCK AND THREAD SIZES HERE!!!!!!!
    if (complex) {
        // TODO THIS SIZE PROBABLY NEEDS TO CHANGE
        clear_complex<<<1, size>>>(samples.complex);
    } else {
        // TODO THIS SIZE PROBABLY NEEDS TO CHANGE
        clear_real<<<1, size>>>(samples.real);
    }
}

void GPUSamples::load(std::vector<std::vector<double>> &source, int start, int end) {
    // if before the loop this time so that we can appropriately allocate an additional buffer to the 
    if (complex) {
        auto host_buf = new hipfftDoubleComplex[end - start];
        // TODO do we need this buffer for both?
        for (int i = 0; i < end - start; i++) {
            host_buf[i].x = source[0][i + start];
            host_buf[i].y = source[1][i + start];
        }

        hipMemcpy(samples.complex, host_buf, sizeof(hipfftDoubleComplex) * (end - start), hipMemcpyHostToDevice);

        delete[] host_buf;
    } else {
        auto host_buf = new hipfftDoubleReal[end - start];
        for (int i = 0; i < end - start; i++) {
            host_buf[i] = source[0][i + start];
        }

        hipMemcpy(samples.real, host_buf, sizeof(hipfftDoubleReal) * (end - start), hipMemcpyHostToDevice);

        delete[] host_buf;
    }
}

// TODO we might want to be more defensive here, in case fftSize != window size
void GPUSamples::applyWindow(thrust::device_vector<double> window) {
    if (complex) {
        apply_window_complex<<<1, size>>>(samples.complex, thrust::raw_pointer_cast(window.data()));
    } else {
        apply_window_real<<<1, size>>>(samples.real, thrust::raw_pointer_cast(window.data()));
    }
}
