#include "hip/hip_runtime.h"
#include "GPUSamples.cuh"

#include <thrust/host_vector.h>

#include "kernel_utils.cuh"

__global__ void clear_real(hipfftDoubleReal* data) {
    auto idx = get_thread_index();
    data[idx] = 0;
}

__global__ void clear_complex(hipfftDoubleComplex* data) {
    auto idx = get_thread_index();
    // storing in register here to potentially improve performance?
    auto sample = data[idx];
    sample.x = 0;
    sample.y = 0;
}

GPUSamples::GPUSamples(bool complex, int fft_size)
{
    this->complex = complex;
    size = fft_size;

    // TODO consider malloc host
    // TODO we probably need to fix fft_size
    if (complex) {
        hipMalloc((void**) &samples.complex, fft_size * sizeof(hipfftDoubleComplex));
    } else {
        hipMalloc((void**) &samples.real, fft_size * sizeof(hipfftDoubleReal));
    }
}

GPUSamples::~GPUSamples()
{
    if (complex) {
        hipFree(samples.complex);
    } else {
        hipFree(samples.real);
    }
}

bool GPUSamples::isComplex() {
    return complex;
}

// TODO we have to do the rest of these

GPUSamples::Samples GPUSamples::getSamples() {
    return samples;
}

hipfftDoubleReal* GPUSamples::getReal() {
    return samples.real;
}

hipfftDoubleComplex* GPUSamples::getComplex() {
    return samples.complex;
}

void GPUSamples::clear() {
    // TODO WE NEED TO TUNE THE BLOCK AND THREAD SIZES HERE!!!!!!!
    if (complex) {
        // TODO THIS SIZE PROBABLY NEEDS TO CHANGE
        clear_complex<<<1, size>>>(samples.complex);
    } else {
        // TODO THIS SIZE PROBABLY NEEDS TO CHANGE
        clear_real<<<1, size>>>(samples.real);
    }
}

// FIXME WE NEED TO FIX THIS! (MOST LIKELY BY CREATING A KERNEL LIKE WE HAD ABOVE)
// TODO are we able to do this? do we need to use thrust?
void GPUSamples::load(std::vector<std::vector<double>> &source, int start, int end) {
    // if before the loop this time so that we can appropriately allocate an additional buffer to the 
    if (complex) {
        auto host_buf = new hipfftDoubleComplex[end - start];
        // TODO do we need this buffer for both?
        for (int i = 0; i < end - start; i++) {
            host_buf[i].x = source[0][i];
            host_buf[i].y = source[1][i];
        }

        hipMemcpy(samples.complex, host_buf, sizeof(hipfftDoubleComplex) * (end - start), hipMemcpyHostToDevice);

        delete[] host_buf;
    } else {
        auto host_buf = new hipfftDoubleReal[end - start];
        for (int i = 0; i < end - start; i++) {
            host_buf[i] = source[0][i];
        }

        hipMemcpy(samples.real, host_buf, sizeof(double) * (end - start), hipMemcpyHostToDevice);

        delete[] host_buf;
    }
}

void GPUSamples::applyWindow(thrust::device_vector<double> window) {
    for (int i = 0; i < window.size(); i++) {
        if (complex) {
            samples.complex[i].x *= window[i];
            samples.complex[i].y *= window[i];
        } else {
            samples.real[i] *= window[i];
        }
    }
}
