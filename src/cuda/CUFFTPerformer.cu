#include "hip/hip_runtime.h"
#include "CUFFTPerformer.cuh"

#include <memory>
#include <string>
#include <cmath>

#include <iostream>

#include <AudioFile.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>

#include "GPUSamples.cuh"
#include "../hann.cuh"

// TODO can we put this in a shared location?
#define MIN_REPLACEMENT -350.0


CUFFTPerformer::CUFFTPerformer(int fft_size, const std::string file)
{
    this->fft_size = fft_size;

    source = AudioFile<double>(file);

    window = hann<thrust::device_vector<double> >(fft_size);

    complex = source.getNumChannels() == 2;

    in_buffer = new GPUSamples(complex, fft_size);

    hipMalloc((void**)&out_buffer, fft_size * sizeof(hipfftDoubleComplex));

    hipfftPlan1d(&plan, fft_size, complex ? HIPFFT_Z2Z : HIPFFT_D2Z, 1);
}

CUFFTPerformer::~CUFFTPerformer()
{
    hipfftDestroy(plan);
    hipFree(out_buffer);
    delete in_buffer;
}

void CUFFTPerformer::performFFT() {
    auto num_samples = source.getNumSamplesPerChannel();

    // TODO is this supposed to be / 2 iff the data is real instead of at all times?
    auto num_cols = (num_samples / (fft_size / 2));


    // TODO this will have to be copied back to the host in some way
    thrust::host_vector<thrust::host_vector<double> > output(num_cols);

    for (int i = 0; i < num_cols; i++) {
        
        // std::cout << i << std::endl;

        auto cur_col = thrust::host_vector<double>(output_fft_size);

        std::cout << "pre-clear" << std::endl;

        in_buffer->clear();

        std::cout << "post-clear" << std::endl;

        auto start = fft_size / 2 * i;
        auto end = std::min(start + fft_size, source.getNumSamplesPerChannel());

        std::cout << "pre-load" << std::endl;

        in_buffer->load(source.samples, start, end);

        in_buffer->applyWindow(window);

        complex ? hipfftExecZ2Z(plan, in_buffer->getComplex(), out_buffer, HIPFFT_FORWARD) : hipfftExecD2Z(plan, in_buffer->getReal(), out_buffer) ;

        // auto out_buf_cast = 

        // TODO this needs to be kernel-ified as well!
        for (int j = 0; j < cur_col.size(); j++) {
            // cur_col[j] = st::abs(out_buf_cast[j]);
            // std::abs(std::)
            // TODO we need to take the abs of the thing that we just computed and go from there
            cur_col[j] = pow(out_buffer[j].x, 2) + pow(out_buffer[j].y, 2);
        }

        // NOTE if using vscode, the error squiggle under `__device__` is a false negative; this code compiles fine!
        thrust::transform(cur_col.begin(), cur_col.end(), cur_col.begin(), [=] __device__ (double x) {
            double logscale = 10.0 * log10(x);
            if (isfinite(logscale)) {
                logscale = MIN_REPLACEMENT;
            }
            return logscale;
        });

        output[i] = cur_col;

    }


    // TODO: WE NEED TO COPY THE DATA BACK ONTO THE HOST IN SOME WAY!!!!!!!!!!!!!!!!


}