#include "CUFFTPerformer.cuh"

#include <memory>

#include "../io/SampleSource.h"
#include "../utils/hann.h"

CUFFTPerformer::CUFFTPerformer(int fft_size, std::shared_ptr<SampleSource> source)
{
    this->fft_size = fft_size;
    this->source = source;
    // the window can simply live on the gpu, and since we only transfer to the
    // host during construction time, it can safely be paged
    hipMalloc((void**) &window, fft_size * sizeof(double));
    auto window_host = hann(fft_size);
    // we need to get the actual underlying raw pointer from the smart pointer
    hipMemcpy(window, window_host.get(), fft_size * sizeof(double), hipMemcpyHostToDevice);

    // both inputs and outputs will likely need to interact with the host, so make these page locked
    hipHostMalloc((void**) &data_buffer, fft_size * sizeof(double));
    hipHostMalloc((void**) &output_buffer, fft_size * sizeof(double));

    // window = new double[fft_size];
    // data_buffer = new double[fft_size];
    // // this internal output buffer is used so that we can have a single plan, but also emit defensive copies of windows when the code gets executed
    // // TODO is sizing correct?
    // output_buffer = new double[fft_size];    
}

CUFFTPerformer::~CUFFTPerformer()
{
    hipFree(output_buffer);
    hipFree(data_buffer);
    hipFree(window);
}
